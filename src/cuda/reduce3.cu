#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

#define NUM_SIZE 1024

__global__ void reduce3(int *g_idata, int *g_odata) {
    __shared__ int sdata[32][32];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[blockIdx.x][tid] = g_idata[i];
    __syncthreads();
    
    for(unsigned int s = blockDim.x / 2; s > 0; s /= 2) {
        if(tid < s) {
            sdata[blockIdx.x][tid] += sdata[blockIdx.x][tid+s];
        }
        __syncthreads();
    }

	g_odata[blockIdx.x] = sdata[blockIdx.x][0];

}


int main() {
    int * a = (int*)malloc(sizeof(int)*NUM_SIZE);
    int * b = (int*)malloc(sizeof(int)*NUM_SIZE);

    for(int i = 0; i < NUM_SIZE; i++) {
        a[i] = i;
    }

    int *a_dev;
    int *b_dev;
    hipSetDevice(0);

    hipMalloc(&a_dev, sizeof(int)*NUM_SIZE);
    hipMalloc(&b_dev, sizeof(int)*NUM_SIZE);
    
    hipMemcpy(a_dev, a, sizeof(int)*NUM_SIZE, hipMemcpyHostToDevice);

    reduce3 <<<32, 32, NUM_SIZE>>> (a_dev, b_dev);
	reduce3 <<<1, 32, 32>>> (b_dev, b_dev);

    hipMemcpy(b, b_dev, sizeof(int)*NUM_SIZE, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
	for(int i = 0; i < 32; i++)
	    printf("%d\n", b[i]);
    
    hipFree(a_dev);
    hipFree(b_dev);

    free(a);
    free(b);
}
