#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

#define NUM_SIZE 1024

template <unsigned int blockSize>
__global__ void reduce5(int *g_idata, int *g_odata) {
	__shared__ volatile int sdata[512];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	sdata[i] = g_idata[i] + g_idata[i + blockSize];
	__syncthreads();

	if (blockSize >= 512) {
		if (tid < 256) {sdata[tid] += sdata[tid + 256];} __syncthreads();
	}
	if (blockSize >= 256) {
		if (tid < 128) {sdata[tid] += sdata[tid + 128];} __syncthreads();
	}
	if (blockSize >= 128) {
		if (tid < 64) {sdata[tid] += sdata[tid + 64];} __syncthreads();
	}
	if (tid < 32) {
		if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
	}
	if (i == 0) {
		g_odata[0] = sdata[0];
	}
}


int main() {
    int * a = (int*)malloc(sizeof(int)*NUM_SIZE);
    int * b = (int*)malloc(sizeof(int)*NUM_SIZE);

    for(int i = 0; i < NUM_SIZE; i++) {
        a[i] = i;
    }

    int *a_dev;
    int *b_dev;
    hipSetDevice(0);

    hipMalloc(&a_dev, sizeof(int)*NUM_SIZE);
    hipMalloc(&b_dev, sizeof(int)*NUM_SIZE);
    
    hipMemcpy(a_dev, a, sizeof(int)*NUM_SIZE, hipMemcpyHostToDevice);

    reduce5<512><<<1, 512, NUM_SIZE>>> (a_dev, b_dev);

    hipMemcpy(b, b_dev, sizeof(int)*NUM_SIZE, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
	for(int i = 0; i < 32; i++)
	     printf("%d\n", b[i]);
    
    hipFree(a_dev);
    hipFree(b_dev);

    free(a);
    free(b);
}
