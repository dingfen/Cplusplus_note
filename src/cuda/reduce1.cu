#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

__global__ void reduce1(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    
    for(unsigned int s = 1; s < blockDim.x; s *= 2) {
        if(tid % (2*s) ==0) {
            sdata[tid] += sdata[tid+s];
        }
        __syncthreads();
    }

    if(tid == 0) {
		g_odata[blockIdx.x] = sdata[0];
    }
}

__global__ void test(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];
    printf("hello world\n");
}

int main() {
    int * a = (int*)malloc(sizeof(int)*32);
    int * b = (int*)malloc(sizeof(int)*32);

    for(int i = 0; i < 32; i++) {
        a[i] = i;
    }

    int *a_dev;
    int *b_dev;
    hipSetDevice(0);

    hipMalloc(&a_dev, sizeof(int)*32);
    hipMalloc(&b_dev, sizeof(int)*32);
    
    hipMemcpy(a_dev, a, sizeof(int)*32, hipMemcpyHostToDevice);

    reduce1 <<<1, 32, 32>>> (a_dev, b_dev);

    hipMemcpy(b, b_dev, sizeof(int)*32, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    printf("%d\n", b[0]);
    
    hipFree(a_dev);
    hipFree(b_dev);

    free(a);
    free(b);
}
